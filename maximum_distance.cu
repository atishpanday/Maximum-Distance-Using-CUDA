#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define N 5

__global__ void calculate_distance(int *X, int *Y) {

	unsigned id1 = ( blockIdx.x * 30 + threadIdx.x ) / N;
	unsigned id2 = ( blockIdx.x * 30 + threadIdx.x % N ) % N + id1 + 1;
	
	if(id1 >= N || id2 >= N) return;
	
	int x1 = X[id1], y1 = Y[id1], x2 = X[id2], y2 = Y[id2];
	
	printf("\nx1 = %d, y1 = %d, x2 = %d, y2 = %d", x1, y1, x2, y2);
	
	float distance = sqrt((float) (x2 - x1)*(x2 - x1) + (y2 - y1)*(y2 - y1));
	
	printf("\ndistance = %f", distance);
}

int main() {
	
	int X[N], Y[N];
	int *dX, *dY;
	time_t t;
	
	srand((unsigned) time(&t));
	
	printf("The points are: \n");
	for(unsigned ii = 0; ii < N; ii++) {
		X[ii] = rand() % 10;
		Y[ii] = rand() % 10;
	}
	
	for(unsigned ii = 0; ii < N; ii++) {
		printf("(%d, %d), ", X[ii], Y[ii]);
	}
	
	hipMalloc(&dX, N * sizeof(int));
	hipMalloc(&dY, N * sizeof(int));
	
	hipMemcpy(dX, X, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dY, Y, N * sizeof(int), hipMemcpyHostToDevice);
	
	int nblocks = ceil((float) (N * (N - 1) / 2 ) / 30);
	
	printf("\nnblocks = %d\n", nblocks);
	
	calculate_distance<<<nblocks, 30>>>(dX, dY);
	hipDeviceSynchronize();
	
	return 0;
}

// lets assume N threads. Then we launch the kernel with N^2 threads.
// Then we launch the kernel with 1024 threads and ceil(float) N*N / 1024) blocks.
// each kernel will have threadIdx.x from 0 to 1023, blockDim = N*N/1024 and blockIdx.x from 0 to N*N/1024 - 1

// we have x1 and x2 ranging from 0 to N each
// we need to find the distance between each pair of points and find the maximum distance.

// the pairs will be:
/*
	(0, 1)
	(0, 2)
	(0, 3)
	.
	.
	.
	.
	(1, 2)
	(1, 3)
	(1, 4)
	.
	.
	.
	.
	and so on.
	
	Each x1 starts with i and each corresponding x2 starts with i + 1 and goes up to N - 1
	
	threadId goes from 0 to 1023 and then block id changes.
	
	we have to combine blockId and threadId to make x1 and x2
	
	assume we have 50 ponts - 50 * 49 / 2 = 1225 pairs
	then we have 2 blocks, each block has 1024 threads from 0 to 1023
	
	the first thread has unique id (0, 0) where first 0 is blockid and second 0 is threadid
	
	x1 = X[threadIdx.x / N] x2 = X[blockIdx.x + threadIdx.x % N + 1];
	
	threadidx from 0 to 49 will give 0 for x1 and from 1 to 50 for x2
	
	then from 50 to 99, x1 = 1 and x2 will go from 1 to 50 again
	
	assume there are 30 threads in each block total
	
	then, from 0 to 29 we get x1 = 0 and x2 = 1 to 30
	
	then, blockid changes to 1 and threadid again goes frm 0 to 29, and x1 now is = 0 again, which does not work, since we need x1 to equal 0 from threadid 0 to 19, then 1 from threadid 20 onwards.
	
	
	
*/




























